#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void sum(int* a, int* b, int* c)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int size = 1024;

    // Allocate memory on CPU
    int* a = (int*)malloc(sizeof(int) * size);
    int* b = (int*)malloc(sizeof(int) * size);
    int* c = (int*)malloc(sizeof(int) * size);

    // Allocate memory on GPU
    int* a_d, *b_d, *c_d;
    hipMalloc((void**)&a_d, sizeof(int) * size);
    hipMalloc((void**)&b_d, sizeof(int) * size);
    hipMalloc((void**)&c_d, sizeof(int) * size);
    // kernal_name<<<num_of_blocks, num_of_threads_per_block>>>();
    // cudaDeviceSynchronize();

    // Fill array with values
    for (int i = 0; i < size; ++i)
    {
        a[i] = i;
        b[i] = size - i;
    }

    // Copy data from CPU to GPU
    hipMemcpy(a_d, a, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, sizeof(int) * size, hipMemcpyHostToDevice);

    // Calculate sum
    sum<<<1, 1024>>>(a_d ,b_d, c_d);
    hipDeviceSynchronize();
    
    // Copy result from GPU tp CPU
    hipMemcpy(c, c_d, sizeof(int) * size, hipMemcpyDeviceToHost);

    // Print result
    printf("Execution completed\n");
    for (int i = 0; i < size; ++i)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }


    // Free memory on GPU
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    // Free memory on CPU
    free(a);
    free(b);
    free(c);
    return 0;
}