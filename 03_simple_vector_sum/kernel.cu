#include "hip/hip_runtime.h"



#include <stdio.h>

__global__ void sum(int* a, int* b, int* c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int size = 2048;

    // Allocate memory on CPU
    int* a = (int*)malloc(sizeof(int) * size);
    int* b = (int*)malloc(sizeof(int) * size);
    int* c = (int*)malloc(sizeof(int) * size);

    // Allocate memory on GPU
    int* a_d, *b_d, *c_d;
    hipMalloc((void**)&a_d, sizeof(int) * size);
    hipMalloc((void**)&b_d, sizeof(int) * size);
    hipMalloc((void**)&c_d, sizeof(int) * size);
    // kernal_name<<<num_of_blocks, num_of_threads_per_block>>>();
    // cudaDeviceSynchronize();

    // Fill array with values
    for (int i = 0; i < size; ++i)
    {
        a[i] = i;
        b[i] = size - i;
    }

    // Copy data from CPU to GPU
    hipMemcpy(a_d, a, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, sizeof(int) * size, hipMemcpyHostToDevice);

    const int grid_num = 64;
    const int block_dim = 32;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Calculate sum
    hipEventRecord(start);
    sum<<<grid_num, block_dim>>>(a_d ,b_d, c_d, size);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0.F;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Copy result from GPU tp CPU
    hipMemcpy(c, c_d, sizeof(int) * size, hipMemcpyDeviceToHost);

    printf("Execution time sum<<<%d,%d>>>(): %f ms\n", grid_num, block_dim, milliseconds);

    // Print result
    // printf("Execution completed\n");
    // for (int i = 0; i < size; ++i)
    // {
    //     printf("%d + %d = %d\n", a[i], b[i], c[i]);
    // }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free memory on GPU
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    // Free memory on CPU
    free(a);
    free(b);
    free(c);
    return 0;
}