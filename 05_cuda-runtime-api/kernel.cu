#include <stdio.h>
#include <hip/hip_runtime_api.h>

int main()
{
    int num_devices;
    hipGetDeviceCount(&num_devices);
    for(int i = 0; i < num_devices; ++i)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device number: %d\n", i);
        printf("Device name: %s\n", prop.name);
        printf("Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        printf("Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8.)/1.0e6);
        printf("Total Global Memory: %f (Gb)\n", prop.totalGlobalMem / 1024. / 1024. / 1024.);
        printf("Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("Number of SMs: %d\n", prop.multiProcessorCount);
        printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("Max threads dimensions: x = %d, y = %d, z = %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max grid dimensions: x = %d, y = %d, z = %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    }
    
    return 0;
}