#include "hip/hip_runtime.h"



#include <stdio.h>


inline void gpuAssert(hipError_t err_code, const char* file, int line, bool abort=true)
{
    if(err_code != hipSuccess)
    {
        fprintf(stderr, "Cuda error: %s %s line %d\n", hipGetErrorString(err_code), file, line);
        if(abort){exit(err_code);}
    }
}
// Error checking macro
#define cudaCheckError(res){ gpuAssert(res, __FILE__, __LINE__); }

inline void gpuKernelAssert(const char* file, int line, bool abort=true)
{
    hipError_t err_code = hipGetLastError();
    if(err_code != hipSuccess)
    {
        fprintf(stderr, "Cuda kernel error: %s %s line %d\n", hipGetErrorString(err_code), file, line);
        if(abort){exit(err_code);}
    }
}
#define cudaKernelCheckError(){ gpuKernelAssert(__FILE__, __LINE__); }


__global__ void sum(int* a, int* b, int* c, long n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const long size = 1024LL * 1024 * 1024 * 20;

    // cudaError_t err;
    
    // Allocate memory on GPU
    int* a_d, *b_d, *c_d;
    cudaCheckError(hipMalloc((void**)&a_d, sizeof(int) * size))    // Check synchronous error
    cudaCheckError(hipMalloc((void**)&b_d, sizeof(int) * size))    // Check synchronous error
    cudaCheckError(hipMalloc((void**)&c_d, sizeof(int) * size))    // Check synchronous error
    // kernal_name<<<num_of_blocks, num_of_threads_per_block>>>();
    // cudaDeviceSynchronize();
    
    // Allocate memory on CPU
    int* a = (int*)malloc(sizeof(int) * size);
    int* b = (int*)malloc(sizeof(int) * size);
    int* c = (int*)malloc(sizeof(int) * size);

    // Fill array with values
    for (int i = 0; i < size; ++i)
    {
        a[i] = i;
        b[i] = size - i;
    }

    // Copy data from CPU to GPU
    hipMemcpy(a_d, a, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, sizeof(int) * size, hipMemcpyHostToDevice);

    const int grid_num = 64;
    const int block_dim = 32;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Calculate sum
    hipEventRecord(start);
    
    sum<<<grid_num, block_dim>>>(a_d ,b_d, c_d, size);
    cudaKernelCheckError()  // Check asynchronous error

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0.F;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Copy result from GPU tp CPU
    hipMemcpy(c, c_d, sizeof(int) * size, hipMemcpyDeviceToHost);

    printf("Execution time sum<<<%d,%d>>>(): %f ms\n", grid_num, block_dim, milliseconds);

    // Print result
    // printf("Execution completed\n");
    // for (int i = 0; i < size; ++i)
    // {
    //     printf("%d + %d = %d\n", a[i], b[i], c[i]);
    // }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free memory on GPU
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    // Free memory on CPU
    free(a);
    free(b);
    free(c);
    return 0;
}